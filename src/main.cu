
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    const int N = 3;
    const int M = 2;
    const int K = 3;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, N * K * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, K * M * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, N * M * sizeof(float), hipHostMallocDefault);

    // Initialize matrices h_A and h_B
    for (int i = 0; i < N * K; i++) {
        h_A[i] = i;
    }

    for (int i = 0; i < K * M; i++) {
        h_B[i] = i;
    }

    // Allocate device memory and copy data to device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * K * sizeof(float));
    hipMalloc(&d_B, K * M * sizeof(float));
    hipMalloc(&d_C, N * M * sizeof(float));

    hipblasSetMatrix(N, K, sizeof(float), h_A, N, d_A, N);
    hipblasSetMatrix(K, M, sizeof(float), h_B, K, d_B, K);

    // Perform matrix multiplication
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_A, N, d_B, K, &beta, d_C, N);

    // Copy the result back to host
    hipblasGetMatrix(N, M, sizeof(float), d_C, N, h_C, N);

    // Display the result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << h_C[i * M + j] << " ";
        }
        std::cout << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    hipblasDestroy(handle);

    return 0;
}